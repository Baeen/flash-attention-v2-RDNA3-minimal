#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bfloat16.h>
#include <hip/amd_detail/amd_hip_bf16.h>
#include <hip/amd_detail/amd_hip_fp16.h>

#include <rocwmma/rocwmma.hpp>

using rocwmma::accumulator;
using rocwmma::col_major;
using rocwmma::matrix_a;
using rocwmma::matrix_b;
using rocwmma::row_major;

using rocwmma::bfloat16_t;
using rocwmma::float16_t;
using rocwmma::float32_t;


const int ROCWMMA_M = 16;
const int ROCWMMA_N = 16;
const int ROCWMMA_K = 16;

const int N_WAVES = 16;
const int WAVE_SIZE = 32;


#define ComputeType float16_t
#define AT_PTR_TYPE at::Half
#define TORCH_DTYPE torch::kFloat16

typedef _Float16 fp16_frag __attribute__((ext_vector_type(16)));
typedef float fp32_frag __attribute__((ext_vector_type(8)));

#define HALF16(pointer) (reinterpret_cast<fp16_frag *>((void *)&(pointer))[0])


__global__ void gemm_kernel(
    float16_t *__restrict__ A,
    float16_t *__restrict__ B, 
    float16_t *__restrict__ C, 
    //float16_t *__restrict__ D,
    int m, int n, int k
)
{

    __shared__ fp16_frag trLds[N_WAVES][16];

    fp16_frag fragA[2];
    fp16_frag fragB[2]; 
    // asm volatile("s_sleep 0");

    const int wave_id = __builtin_amdgcn_readfirstlane(threadIdx.x / WAVE_SIZE);
    const int lane_id = threadIdx.x % WAVE_SIZE;
    const int wmma_lane = (threadIdx.x % 16);

    for (int wave_off = 0; wave_off < ((m * n) / (ROCWMMA_M * ROCWMMA_N) + N_WAVES - 1) / N_WAVES; wave_off++)
    {
        int wave_xy = __builtin_amdgcn_readfirstlane(wave_id + wave_off * N_WAVES);

        int wave_x = __builtin_amdgcn_readfirstlane(wave_xy % (n / ROCWMMA_N));
        int wave_y = __builtin_amdgcn_readfirstlane(wave_xy / (n / ROCWMMA_N));

        int blk_x = __builtin_amdgcn_readfirstlane(wave_x * ROCWMMA_N);
        int blk_y = __builtin_amdgcn_readfirstlane(wave_y * ROCWMMA_M);
        if ((blk_x < n) && (blk_y < m))
        {

            fp32_frag fragACC;

// #pragma unroll
            for (int ele = 0; ele < 8; ++ele)
            {
                const int r = ele * 2 + (lane_id / 16);
                fragACC[ele] = (C + (blk_y * n + blk_x))[r * n + wmma_lane];
            }

            for (int i = 0; i < k; i += ROCWMMA_K*2)
            {
                // for(int ele = 0; ele < 16; ele++)
                // {
                //     fragA[ele] = (A + (blk_y * k + i))[wmma_lane * k + ele]; //lda = k
                //     //fragB[ele] = (B + (i * n + blk_x))[ele * n + wmma_lane]; // A @ B 
                //     fragB[ele] = (B + (blk_x * k + i))[wmma_lane * k + ele];   // A @ B^T

                // }

                //fragA = (A + (blk_y * k + i))[wmma_lane * k + ele];

                fragA[0] = HALF16((A + (blk_y * k + i))[wmma_lane * k]);

                trLds[wave_id][wmma_lane] = HALF16((B + (i * n + blk_x))[wmma_lane * n]);
                for(int ele = 0; ele < 16; ele++)
                    fragB[0][ele] = ((float16_t *)&trLds[wave_id])[ele * 16 + wmma_lane];

                fragA[1] = HALF16((A + (blk_y * k + i + ROCWMMA_K))[wmma_lane * k]);

                trLds[wave_id][wmma_lane] = HALF16((B + ((i + ROCWMMA_K) * n + blk_x))[wmma_lane * n]);
                for(int ele = 0; ele < 16; ele++)
                    fragB[1][ele] = ((float16_t *)&trLds[wave_id])[ele * 16 + wmma_lane];

                asm volatile("v_wmma_f32_16x16x16_f16 %0, %1, %2, %0" : "=v"(fragACC) : "v"(fragA[0]), "v"(fragB[0]), "0"(fragACC));
                asm volatile("v_wmma_f32_16x16x16_f16 %0, %1, %2, %0" : "=v"(fragACC) : "v"(fragA[1]), "v"(fragB[1]), "0"(fragACC));

                // fragACC = __builtin_amdgcn_wmma_f32_16x16x16_f16_w32(fragA[0], fragB[0], fragACC);
                // fragACC = __builtin_amdgcn_wmma_f32_16x16x16_f16_w32(fragA[1], fragB[1], fragACC);
                // asm volatile(
                //     "v_wmma_f32_16x16x16_f16 %0, %1, %2, %0 \n"
                //     "v_wmma_f32_16x16x16_f16 %0, %3, %4, %0 \n"
                // : "=v"(fragACC) : 
                // "v"(fragA[0]), "v"(fragB[0]), "v"(fragA[1]), "v"(fragB[1]), 
                // "0"(fragACC));


                // asm volatile("v_wmma_f32_16x16x16_f16 %0, %1, %2, %0" : "=v"(fragACC) : "v"(fragA[1]), "v"(fragB[1]), "0"(fragACC));

                // fragA[2] = HALF16((A + (blk_y * k + i + 2*ROCWMMA_K))[wmma_lane * k]);
                // fragB[2] = HALF16((B + (blk_x * k + i + 2*ROCWMMA_K))[wmma_lane * k]);

                // fragA[3] = HALF16((A + (blk_y * k + i + 3*ROCWMMA_K))[wmma_lane * k]);
                // fragB[3] = HALF16((B + (blk_x * k + i + 3*ROCWMMA_K))[wmma_lane * k]);

                // fragACC = __builtin_amdgcn_wmma_f32_16x16x16_f16_w32(fragA[2], fragB[2], fragACC);
                // fragACC = __builtin_amdgcn_wmma_f32_16x16x16_f16_w32(fragA[3], fragB[3], fragACC);
                

            }
            __syncthreads();

#pragma unroll
            for (int ele = 0; ele < 8; ++ele)
            {
                const int r = ele * 2 + (lane_id / 16);
                (C + (blk_y * n + blk_x))[r * n + wmma_lane] = fragACC[ele];
            }

        }
    }
    __syncthreads();

    // asm volatile("s_sleep 0");

}


torch::Tensor forward(
    torch::Tensor A, 
    torch::Tensor B, 
    torch::Tensor C, 
    int m, int n, int k
)
{

    //auto optD = torch::TensorOptions().dtype(torch::kFloat16).device(torch::kCUDA);
    //auto D = torch::zeros({m, n}, optD);

    auto gridDim = dim3(1, 1, 1);
    auto blockDim = dim3(WAVE_SIZE * N_WAVES);
    gemm_kernel<<<gridDim, blockDim, 0>>>(
        (ComputeType *)A.data_ptr<AT_PTR_TYPE>(),
        (ComputeType *)B.data_ptr<AT_PTR_TYPE>(),
        (ComputeType *)C.data_ptr<AT_PTR_TYPE>(), 
        m,n,k
    );

    return C;
}
